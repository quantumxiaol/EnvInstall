
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    int a[] = {1, 2, 3};
    int b[] = {4, 5, 6};
    int c[3];
    int n = 3;

    int *d_a, *d_b, *d_c;

    // 分配显存
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    // 拷贝数据到显存
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // 在 GPU 上执行 kernel
    vectorAdd<<<1, 3>>>(d_a, d_b, d_c, n);

    // 拷贝结果回主机内存
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // 释放显存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}