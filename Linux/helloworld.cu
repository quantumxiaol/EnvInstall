
#include <hip/hip_runtime.h>
#include <stdio.h>
// 核函数必须使用 __global__ 修饰符声明。
__global__ void say_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");

    // 启动内核，核函数调用语法
    say_hello<<<1, 1>>>();
    // 启动一个网格（grid），包含 1 个线程块（block）。
    // 每个线程块中包含 1 个线程（thread）。
    // 所以总共会启动 1 个线程 来执行 say_hello 这个核函数。

    // 等待GPU完成
    hipDeviceSynchronize();

    return 0;
}

// kernel_name<<<grid_dim, block_dim, shared_mem_size, stream>>>(args...);
// 其中：
// kernel_name：你要调用的 CUDA 核函数名。
// <<<...>>>：这是 CUDA 特有的语法，用于配置核函数执行的线程结构和参数。
// grid_dim：网格中线程块的数量（dim3 类型），决定启动多少个线程块。
// block_dim：每个线程块中线程的数量（dim3 类型）。
// shared_mem_size（可选）：为该核函数分配的共享内存大小（以字节为单位），默认为 0。
// stream（可选）：指定异步流，默认为 0（即默认流）。

// CUDA 的 <<<...>>> 语法是扩展的 C++ 语法，只能被 CUDA 编译器识别（如 nvcc）。