
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void say_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");

    // 启动内核
    say_hello<<<1, 1>>>();

    // 等待GPU完成
    hipDeviceSynchronize();

    return 0;
}